
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream> // file input
#include <string> 
#include <vector> // list
#include <unistd.h> // getopt
#include <stdlib.h> // exit

using namespace std;

// global var
int i, j, k; // ITERATE VARIABLES

// cuda kernel
__global__ void nextBoard(int x, int y, int* board, int* boardR)
{     
    int boardSize = x * y;
    int idx = blockDim.x*blockIdx.x+threadIdx.x;
    if(idx >= boardSize) return; // handles index range error causing memcheck errors

    int posX = idx % x;    // x coordinate
    int posY = idx / x;    // y coordinate

    int leftX = (posX + x - 1) % x;   // one left of idx
    int rightX = (posX + 1) % x;      // one right of idx
    int posYUp = (posY + y - 1) % y;   // one up of idx
    int posYDown = (posY + 1) % y;     // one down of idx

    // Alive neighbours for each point idx
    int neighbours = board[leftX + posYUp*x] 
        + board[posX + posYUp*x] 
        + board[rightX + posYUp*x]
        + board[leftX + posY*x] 
        + board[rightX + posY*x]
        + board[leftX + posYDown*x] 
        + board[posX + posYDown*x] 
        + board[rightX + posYDown*x];

    // Assigning new cell value
    boardR[posX+posY*x] = (neighbours == 3 || (neighbours == 2 && board[posY * x + posX])) ? 1 : 0;
}

int main(int argc, char **argv)
{
    int x, y;                       // board dimensions
    int *d_board, *d_boardR;        // int array pointers
    char* filename = argv[argc-1];  // filename from command line, last argument
    string line;                    // str line to extract from file
    vector<string> vec;             // vec vec to extract board dimensions
    int option;                     // getopt var
    int iter = 1;                   // board iteration variable
    bool verbose = false;       

    // getopt - iterations and verbose
    while((option = getopt(argc, argv, "i:v"))!=-1){
        switch (option) {
            case 'i' :
                iter = atoi(optarg);
                // printf("%d", iter);
                break;
            case 'v' : 
                verbose = true;
                // printf("verbose");
                break;
            default : 
                printf("you broke it");       
        }
    }

    // Get input file, convert into vector, could be replace with a function
    ifstream infile(filename);
    while (!infile.eof()){
        getline(infile, line);
        if(!line.empty()){
            vec.push_back(line);
        }
        // printf("%s\n",line.c_str());
    }
    infile.close();

    // Use vector size for board dimensions
    y = vec.size();
    x = vec.front().size();
    
    int board[x*y], boardR[x*y];   
    // printf("height: %d | width: %d\n", y, x);

    // Vector List to Single Dimension Array Conversion | -,X replaced with 0,1
    for(i=0; i<y; i++){
        for(j=0; j<x; j++){
            if(vec[i][j] == '-'){
                board[i*x+j]=0;
                boardR[i*x+j]=0;
            }
            else if(vec[i][j] == 'X'){ 
                board[i*x+j]=1; 
                boardR[i*x+j]=1;
            }
        }
    }

    // Allocate device memory for board arrays
    hipMalloc((void **)&d_board, sizeof(int)*x*y);
    hipMalloc((void **)&d_boardR, sizeof(int)*x*y);

    // Copy host arrays to device 
    hipMemcpy(d_board, board, sizeof(int)*x*y, hipMemcpyHostToDevice);
    hipMemcpy(d_boardR, boardR, sizeof(int)*x*y, hipMemcpyHostToDevice);

    // for calculating cuda blocks, board size/threads for blocks needed
    const int xy = 1 + ((x*y-1)/32);

    // Pick number of iterations to run board on GPU
    for(i=0; i<iter; i++){
        nextBoard<<<xy,32>>>(x, y, d_board, d_boardR); // 1024 threads per block
        hipDeviceSynchronize();
        // Swapping CUDA Kernel input board
        int *temp = d_board;
        d_board = d_boardR;
        d_boardR = temp;
        // if -v, then print each iteration

        if(verbose==true){
            // Animate - Clear terminal
            printf("\033[2J\033[H");
            usleep(10000);

            hipMemcpy(board, d_board, sizeof(int)*x*y, hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            for(j=0; j<y; j++){
                for(k=0; k<x; k++){
                    if(board[j*x+k]== 0) cout << '-';
                    else if(board[j*x+k]== 1 ) cout << 'X';         
                }
                cout << '\n';
            }
            cout << '\n';
        }
        
    }
    // Copy board back from device memory after iterations
    hipMemcpy(boardR, d_board, sizeof(int)*x*y, hipMemcpyDeviceToHost);

    // Free memory
    hipFree(d_board);
    hipFree(d_boardR);

    // Print final board to console
    if(verbose==false){
        for(i=0; i<y; i++){
            for(j=0; j<x; j++){
                if(boardR[i*x+j]== 0) cout << '-';
                else if(boardR[i*x+j]== 1 ) cout << 'X';         
            }
            cout << '\n';
        }
    }
}