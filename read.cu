
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream> // file input
#include <string> 
#include <vector> // list
#include <unistd.h> // getopt
#include <stdlib.h> // exit

using namespace std;

// global var
int i, j, k; // ITERATE VARIABLES

// cuda kernel
__global__ void nextBoard(int x, int y, int* board, int* boardR)
{     
    int boardSize = x * y;
    int idx = blockDim.x*blockIdx.x+threadIdx.x;

    if(idx >= boardSize) return; // handles index range error
    
    // x, y coordinates
    int posX = idx % x;    // x coordinate
    int posY = idx / x;    // y coordinate

    int leftX = (posX + x - 1) % x;   // one left of idx
    int rightX = (posX + 1) % x;      // one right of idx

    int posYUp = (posY + y - 1) % y;   // one up of idx
    int posYDown = (posY + 1) % y;     // one down of idx

    // TEST - no values exceed board size
    // if((posX > boardSize|| posY > boardSize || leftX > boardSize || rightX > boardSize || posYUp > boardSize || posYDown > boardSize)) printf("Error");

    // Alive neighbours for each point idx
    int neighbours = board[leftX + posYUp*x] 
        + board[posX + posYUp*x] 
        + board[rightX + posYUp*x]
        + board[leftX + posY*x] 
        + board[rightX + posY*x]
        + board[leftX + posYDown*x] 
        + board[posX + posYDown*x] 
        + board[rightX + posYDown*x];

    
    // Assigning new cell value
    boardR[posX+posY*x] = (neighbours == 3 || (neighbours == 2 && board[posY * x + posX])) ? 1 : 0;
}

int main(int argc, char **argv)
{
    int x, y;                       // BOARD DIMENSIONS
    int *d_board, *d_boardR;        // ARRAY POINTERS
    char* filename = argv[argc-1];  // filename from command line, last argument
    string line;                    // str line to extract from file
    vector<string> vec;             // vec vec to extract BOARD DIMENSIONS
    int option;                     // getopt var
    int iter = 1;                   // board iteration variable
    bool verbose = false;       

    // getopt - iterations and verbose
    while((option = getopt(argc, argv, "i:v"))!=-1){
        switch (option) {
            case 'i' :
                iter = atoi(optarg);
                // printf("%d", iter);
                break;
            case 'v' : 
                verbose = true;
                // printf("verbose");
                break;
            default : 
                printf("you broke it");       
        }
    }

    // Get input file, convert into vector, could be replace with a function
    ifstream infile(filename);
    while (!infile.eof()){
        getline(infile, line);
        vec.push_back(line);
    }
    infile.close();

    // Use vector size for board dimensions
    y = vec.size();
    x = vec.front().size();

    int board[x*y], boardR[x*y];

    // Vector List to Single Dimension Array Conversion | -,X replaced with 0,1
    for(i=0; i<x; i++){
        for(j=0; j<y; j++){
            if(vec[i][j] == '-'){
                board[i*x+j]=0;
                boardR[i*x+j]=0;
            }
            else if(vec[i][j] == 'X'){ 
                board[i*x+j]=1; 
                boardR[i*x+j]=1;
            }
            else cout << "Your input contains invalid characters";
        }
    }

    // Allocate device memory for board arrays
    hipMalloc((void **)&d_board, sizeof(int)*x*y);
    hipMalloc((void **)&d_boardR, sizeof(int)*x*y);

    // Copy host arrays to device 
    hipMemcpy(d_board, board, sizeof(int)*x*y, hipMemcpyHostToDevice);
    hipMemcpy(d_boardR, boardR, sizeof(int)*x*y, hipMemcpyHostToDevice);

    // for calculating cuda blocks, board size/threads for blocks needed
    const int xy = 1 + ((x*y-1)/32);

    // Pick number of iterations to run board on GPU
    for(i=0; i<iter; i++){
        nextBoard<<<xy,32>>>(x, y, d_board, d_boardR); // 32 threads per block
        hipDeviceSynchronize();
        // Swapping CUDA Kernel input board
        int *temp = d_board;
        d_board = d_boardR;
        d_boardR = temp;
        
        // if -v, then print each iteration
        if(verbose==true){
            hipMemcpy(board, d_board, sizeof(int)*x*y, hipMemcpyDeviceToHost);
            for(j=0; j<x; j++){
                for(k=0; k<y; k++){
                    if(board[j*x+k]== 0) cout << '-';
                    else if(board[j*x+k]== 1 ) cout << 'X';         
                }
                cout << '\n';
            }
        }
        cout << '\n';
    }
    // Copy board back from device memory after iter
    hipMemcpy(boardR, d_boardR, sizeof(int)*x*y, hipMemcpyDeviceToHost);

    // Free memory
    hipFree(d_board);
    hipFree(d_boardR);

    // Print final board to console
    if(verbose==false){
        for(i=0; i<x; i++){
            for(j=0; j<y; j++){
                if(boardR[i*x+j]== 0) cout << '-';
                else if(boardR[i*x+j]== 1 ) cout << 'X';         
            }
            cout << '\n';
        }
    }
}